#include "linAlg/matrix.h"



Matrix<float>& matMul(hipblasHandle_t handle, const Matrix<float>& A, const Matrix<float>& B, Matrix<float>& out, bool transA, bool transB) {
	auto opA = transA?HIPBLAS_OP_T:HIPBLAS_OP_N;
	auto opB = transB?HIPBLAS_OP_T:HIPBLAS_OP_N;
	float alpha = 1.0f;
	float beta = 0.0f;
	auto err = hipblasSgemm(handle, opA, opB, transA?A.getM():A.getN(), transB?B.getN():B.getM(), transA?A.getN():A.getM(), 
			&alpha, A.getData(), A.getN(), B.getData(), B.getN(), &beta, out.getData(), out.getN());
	hipDeviceSynchronize();
	if (err) {
		std::cout << "Mat mul error: " << err << std::endl;
		throw err;
	}
	return out;
}

Matrix<float>& matMul_Add(hipblasHandle_t handle, const Matrix<float>& A, const Matrix<float>& B, const Matrix<float>& C, Matrix<float>& out, bool transA, bool transB) {
	float alpha = 1.0f;
	auto opA = transA?HIPBLAS_OP_T:HIPBLAS_OP_N;
	auto opB = transB?HIPBLAS_OP_T:HIPBLAS_OP_N;
	out.gpuSetValues(C.getData());
	auto err = hipblasSgemm(handle, opA, opB, transA?A.getM():A.getN(), transB?B.getN():B.getM(), transA?A.getN():A.getM(), 
			&alpha, A.getData(), A.getN(), B.getData(), B.getN(), &alpha, out.getData(), out.getN());
	hipDeviceSynchronize();
	if (err) {
		std::cout << "Mat mul add error: " << err << std::endl;
		throw err;
	}
	return out;
}

Matrix<float>& add(hipblasHandle_t handle, const Matrix<float>& A, Matrix<float>& out, float alpha) {
	auto err = hipblasSaxpy(handle, A.getN()*A.getM(), &alpha, A.getData(), 1, out.getData(), 1);
	hipDeviceSynchronize();
	if (err) {
		std::cout << "add error: " << err << std::endl;
		throw err;
	}
}
