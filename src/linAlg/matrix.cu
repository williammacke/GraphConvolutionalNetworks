#include "linAlg/matrix.h"



Matrix<float>& matMul(hipblasHandle_t handle, const Matrix<float>& A, const Matrix<float>& B, Matrix<float>& out, bool transA, bool transB) {
	auto opA = transA?HIPBLAS_OP_T:HIPBLAS_OP_N;
	auto opB = transB?HIPBLAS_OP_T:HIPBLAS_OP_N;
	float alpha = 1.0f;
	float beta = 0.0f;
	hipblasSgemm(handle, opA, opB, transA?A.getM():A.getN(), transB?B.getN():B.getM(), transA?A.getN():A.getM(), 
			&alpha, A.getData(), transA?A.getM():A.getN(), B.getData(), transB?B.getM():B.getN(), &beta, out.getData(), out.getN());
	return out;
}

Matrix<float>& matMul_Add(hipblasHandle_t handle, const Matrix<float>& A, const Matrix<float>& B, const Matrix<float>& C, Matrix<float>& out, bool transA, bool transB) {
	float alpha = 1.0f;
	auto opA = transA?HIPBLAS_OP_T:HIPBLAS_OP_N;
	auto opB = transB?HIPBLAS_OP_T:HIPBLAS_OP_N;
	out.gpuSetValues(C.getData());
	hipblasSgemm(handle, opA, opB, transA?A.getM():A.getN(), transB?B.getN():B.getM(), transA?A.getN():A.getM(), 
			&alpha, A.getData(), transA?A.getM():A.getN(), B.getData(), transB?B.getM():B.getN(), &alpha, out.getData(), out.getN());
	return out;
}

Matrix<float>& add(hipblasHandle_t handle, const Matrix<float>& A, Matrix<float>& out, float alpha) {
	hipblasSaxpy(handle, A.getN()*A.getM(), &alpha, A.getData(), 1, out.getData(), 1);
}
