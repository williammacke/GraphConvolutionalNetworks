#include "linAlg/matrix.h"



Matrix<float>& matMul(hipblasHandle_t handle, const Matrix<float>& A, const Matrix<float>& B, Matrix<float>& out) {
	float alpha = 1.0f;
	float beta = 0.0f;
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, A.getN(), B.getM(), A.getM(), 
			&alpha, A.getData(), A.getN(), B.getData(), B.getN(), &beta, out.getData(), out.getN());
	return out;
}

Matrix<float>& matMul_Add(hipblasHandle_t handle, const Matrix<float>& A, const Matrix<float>& B, const Matrix<float>& C, Matrix<float>& out) {
	float alpha = 1.0f;
	out.gpuSetValues(C.getData());
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, A.getN(), B.getM(), A.getM(), 
			&alpha, A.getData(), A.getN(), B.getData(), B.getN(), &alpha, out.getData(), out.getN());
	return out;
}

Matrix<float>& add(hipblasHandle_t handle, const Matrix<float>& A, Matrix<float>& out, float alpha) {
	hipblasSaxpy(handle, A.getN()*A.getM(), &alpha, A.getData(), 1, out.getData(), 1);
}
