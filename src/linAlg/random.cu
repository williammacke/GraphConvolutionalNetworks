#include "hip/hip_runtime.h"
#include "linAlg/random.h"


__global__ void initCurand(hiprandState *state, unsigned long seed, size_t n) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n) {
		hiprand_init(seed, idx, 0, &state[idx]);
	}
}
