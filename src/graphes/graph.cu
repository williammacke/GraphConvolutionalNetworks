#include "graphes/graph.h"


Matrix<float>& sparseMatMul(hipsparseHandle_t handle, const Graph<float>& A, const Matrix<float>& B, Matrix<float>& out) {
	float alpha = 1.0f;
	float beta = 0.0f;
	hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, A.getNumNodes(), A.getNumNodes(),
			A.getNumNodes(), A.getNNZB(), &alpha, A.getDescr(), A.getData(),
			A.getRowInd(), A.getColInd(), A.getBlockDim(), B.getData(), B.getN(), &beta, 
			out.getData(), out.getN());
	auto err = hipDeviceSynchronize();
	if (err) {
		std::cout << "graph mult error: " << err << std::endl;
		throw err;
	}
	return out;
}
