#include "linAlg/matrix.h"
#include <iostream>

struct addTwo {
	__host__ __device__
	float operator() (float a) const{
		return a+2.0f;
	}
};

int main() {
	Matrix<float> a(2,2);
	Matrix<float> b(2, 2);
	Matrix<float> c(2, 2);
	Matrix<float> d(2, 2);

	float data[] = {2,1,1,1};
	float data2[] = {0,0,0,0};
	a.setValues(data);
	b.setValues(data);
	c.setValues(data2);
	float blargh = 0.0f;
	hipMemcpy(&blargh, Matrix<float>::one, sizeof(float),hipMemcpyDeviceToHost );
	std::cout << blargh <<std::endl;


	hipMemcpy(data2, a.getData(), 4*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << data2[0] << data2[1] << data2[2] << data2[3] << std::endl;
	data2[0] = 0;
	hipMemcpy(data2, b.getData(), 4*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << data2[0] << data2[1] << data2[2] << data2[3] << std::endl;
	hipMemcpy(data2, c.getData(), 4*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << data2[0] << data2[1] << data2[2] << data2[3] << std::endl;
	std::cout << Matrix<float>::one << std::endl;
	


	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipError_t error;

	status = hipblasCreate(&handle);
	if (status != hipSuccess) {
		std::cout << "error";
	}

	matMul(handle, a, b, d);
	hipMemcpy(data2, d.getData(), 4*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << data2[0] << data2[1] << data2[2] << data2[3] << std::endl;


	matMul_Add(handle, a, b, c, d);
	hipMemcpy(data2, d.getData(), 4*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << data2[0] << data2[1] << data2[2] << data2[3] << std::endl;


	add(handle, a, d, 0.5f);
	//hipMemcpy(data2, d.getData(), 4*sizeof(float), hipMemcpyDeviceToHost);
	error = hipMemcpy(data2, d.getData(), 4*sizeof(float), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cout << "error" << " " << error << std::endl;
	}
	std::cout << data2[0] << data2[1] << data2[2] << data2[3] << std::endl;



	matElementMul(a, b, d);
	hipMemcpy(data2, d.getData(), 4*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << data2[0] << data2[1] << data2[2] << data2[3] << std::endl;

	matApply(a, d, addTwo{});
	error = hipMemcpy(data2, d.getData(), 4*sizeof(float), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cout << "error" << " " << error << std::endl;
		if (error == hipErrorInvalidValue) {
			std::cout << "invalid value" << std::endl;
		}

		if (error == hipErrorInvalidValue) {
			std::cout << "invalid value" << std::endl;
		}

		if (error == hipErrorInvalidMemcpyDirection) {
			std::cout << "invalid direction" << std::endl;
		}
	}
	std::cout << data2[0] << data2[1] << data2[2] << data2[3] << std::endl;




	hipblasDestroy(handle);
	return 0;
}
