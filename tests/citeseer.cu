#include "layers/graph_conv.h"
#include "layers/network.h"
#include "functions/activations.h"
#include "initializer/random_normal.h"
#include "functions/loss.h"
#include "optimization/gradient_descent.h"
#include <fstream>
#include <string>
#include <unordered_map>
#include <unordered_set>
#include <iostream>

int main() {
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipsparseHandle_t sparseHandle;
	hipsparseCreate(&sparseHandle); 
	std::ifstream content("../data/citeseer/citeseer.content");
	const int numPapers = 3312;
	const int numWords = 3703;
	std::vector<std::string> ids(numPapers);
	std::unordered_map<std::string, size_t> id_map;
	std::string label;
	std::unordered_map<std::string, int> label_map = {
		{"Agents", 0},
		{"AI", 1},
		{"DB", 2},
		{"IR", 3},
		{"ML", 4},
		{"HCI", 5}
	};

	float* data = new float[numPapers*numWords];
	float* labels = new float[numPapers*6];
	for (int i = 0; i < numPapers*6; ++i) {
		labels[i] = 0;
	}


	for (int i = 0; i < numPapers; ++i) {
		content >> ids[i];
		id_map[ids[i]] = i;
		for (int j = 0; j < numWords; ++j) {
			content >> data[j*numPapers+i];
		}
		content >> label;
		int li = label_map[label];
		labels[li*numPapers+i] = 1.0f;
	}

	content.close();
	std::vector<std::vector<size_t>> adj_list(numPapers);
	std::vector<std::unordered_set<size_t>> adj_set(numPapers);
	std::ifstream cites("../data/citeseer/citeseer.cites");


	std::string id1;
	std::string id2;
	for (int i = 0; i < numPapers; ++i) {
		cites >> id1;
		cites >> id2;

		int i1 = id_map[id1];
		int i2 = id_map[id2];

		if (adj_set[i1].find(i2) == adj_set[i1].end()) {
			adj_list[i1].push_back(i2);
			adj_set[i1].insert(i2);
		}
		if (adj_set[i2].find(i1) == adj_set[i2].end()) {
			adj_list[i2].push_back(i1);
			adj_set[i2].insert(i1);
		}
	}

	cites.close();


	Matrix<float> features(numPapers, numWords);
	features.setValues(data);

	Graph<float> g(adj_list, sparseHandle);
	std::cin.get();

	GCNLayer<random_normal_init, relu> layer1("l1", numPapers, numWords, 100, relu(),
			random_normal_init(0, 0.1));
	GCNLayer<random_normal_init, softmax> layer2("l2", numPapers, 100, 6, softmax(),
			random_normal_init(0, 0.1));


	Network<cross_entropy_with_logits, gradient_descent_optimizer, GCNLayer<random_normal_init, relu>, GCNLayer<random_normal_init, softmax>> network(numPapers, 6, {}, gradient_descent_optimizer(0.0001f), handle, sparseHandle, layer1, layer2);
	network.setGraph(&g);
	network.setLabels(labels);

	float* result = new float[numPapers*6];
	try {
	hipMemcpy(result, network.result(features).getData(), sizeof(float)*6*numPapers, hipMemcpyDeviceToHost);
	} catch(int i) { std::cout << "Error " << i << std::endl; }
	for (int i = 0; i < numPapers; ++i) {
		std::cout << ids[i] << " ";
		for (int j = 0; j < 6; j++) {
			std::cout << result[j*numPapers+i] << " ";
		}
		std::cout << std::endl;

	}
	for (int i = 0; i < 10; ++i) {
		std::cout << network.getLoss() << std::endl;
		std::cin.get();
		network.train(10, features);
	}

	delete[] data;
	delete[] labels;
	delete[] result;
	hipblasDestroy(handle);
	hipsparseDestroy(sparseHandle);
}
